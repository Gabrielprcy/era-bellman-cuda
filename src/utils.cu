#include "common.cuh"
#include "utils.cuh"

#pragma clang diagnostic push
#pragma ide diagnostic ignored "misc-no-recursion"
hipError_t sync_device_streams(const int *const device_ids, const hipStream_t *streams, const unsigned log_devices_count) {
  if (log_devices_count == 0)
    return hipSuccess;
  const unsigned log_offset = log_devices_count - 1;
  const unsigned offset = 1 << log_offset;
  HANDLE_CUDA_ERROR(sync_device_streams(device_ids, streams, log_offset));
  HANDLE_CUDA_ERROR(sync_device_streams(device_ids + offset, streams + offset, log_offset));
  device_guard guard;
  for (unsigned i = 0; i < offset; i++) {
    unsigned indexes[] = {i, i + offset};
    for (unsigned j = 0; j < 2; j++) {
      hipEvent_t event;
      HANDLE_CUDA_ERROR(guard.set(device_ids[indexes[j]]));
      HANDLE_CUDA_ERROR(hipEventCreateWithFlags(&event, hipEventDisableTiming));
      HANDLE_CUDA_ERROR(hipEventRecord(event, streams[indexes[j]]));
      HANDLE_CUDA_ERROR(hipStreamWaitEvent(streams[indexes[1 - j]], event));
      HANDLE_CUDA_ERROR(hipEventDestroy(event));
    }
  }
  HANDLE_CUDA_ERROR(guard.reset());
  return hipSuccess;
}
#pragma clang diagnostic pop
