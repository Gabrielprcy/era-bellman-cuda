#include "hip/hip_runtime.h"
#include "common.cuh"
#include "ec.cuh"
#include "msm_kernels.cuh"

namespace msm {

#define MAX_THREADS 128
#define UINT4_COUNT (sizeof(storage) / sizeof(uint4))
#ifndef __CUDACC_DEBUG__
__launch_bounds__(MAX_THREADS)
#endif
    __global__ void initialize_buckets_kernel(point_xyzz *buckets, const unsigned count) {
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= count)
    return;
  const auto bucket_index = gid / UINT4_COUNT;
  const auto element_index = gid % UINT4_COUNT;
  auto elements = reinterpret_cast<uint4 *>(&buckets[bucket_index].zz);
  memory::store<uint4, memory::st_modifier::cs>(elements + element_index, {});
}

__host__ hipError_t initialize_buckets(point_xyzz *buckets, const unsigned count, hipStream_t stream) {
  auto count_u4 = UINT4_COUNT * count;
  const dim3 block_dim = count_u4 < MAX_THREADS ? count_u4 : MAX_THREADS;
  const dim3 grid_dim = (count_u4 - 1) / block_dim.x + 1;
  initialize_buckets_kernel<<<grid_dim, block_dim, 0, stream>>>(buckets, count_u4);
  return hipGetLastError();
}
#undef UINT4_COUNT
#undef MAX_THREADS

#define MAX_THREADS 128
#ifndef __CUDACC_DEBUG__
__launch_bounds__(MAX_THREADS)
#endif
    __global__ void compute_bucket_indexes_kernel(const fd_q::storage *__restrict__ scalars, const unsigned windows_count, const unsigned window_bits,
                                                  unsigned *__restrict__ bucket_indexes, unsigned *__restrict__ base_indexes, const unsigned count) {
  constexpr unsigned highest_bit_mask = 0x80000000;
  const unsigned scalar_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (scalar_index >= count)
    return;
  const unsigned top_window_unused_bits = windows_count * window_bits - fd_q::MBC;
  const unsigned top_window_unused_mask = (1 << top_window_unused_bits) - 1;
  const unsigned top_window_used_bits = window_bits - top_window_unused_bits;
  const unsigned signed_window_bits = window_bits - 1;
  const unsigned top_window_signed_window_bits = top_window_used_bits - 1;
  const unsigned top_bucket_index = 1 << signed_window_bits;
  const unsigned top_window_top_bucket_index = 1 << top_window_signed_window_bits;
  const storage pos_scalar = fd_q::from_montgomery(memory::load(scalars + scalar_index));
  const storage neg_scalar = fd_q::neg(pos_scalar);
  unsigned global_sign = fd_q::lt(pos_scalar, neg_scalar) ? 0 : highest_bit_mask;
  const storage scalar = global_sign ? neg_scalar : pos_scalar;
  unsigned borrow = 0;
  for (unsigned i = 0; i < windows_count; i++) {
    const unsigned window_index = i;
    const unsigned window_mask = window_index << window_bits;
    unsigned bucket_index = fd_q::extract_bits(scalar, window_index * window_bits, window_bits);
    bucket_index += borrow;
    borrow = 0;
    unsigned sign = global_sign;
    if (bucket_index > top_bucket_index) {
      bucket_index = (top_bucket_index << 1) - bucket_index;
      borrow = 1;
      sign ^= highest_bit_mask;
    }
    bool is_top_window = window_index == windows_count - 1;
    const unsigned zero_mask = bucket_index ? 0 : highest_bit_mask;
    bucket_index = ((bucket_index & ((is_top_window ? top_window_top_bucket_index : top_bucket_index) - 1)) << 1) |
                   (bucket_index >> (is_top_window ? top_window_signed_window_bits : signed_window_bits));
    const unsigned bucket_index_offset = is_top_window ? (scalar_index & top_window_unused_mask) << top_window_used_bits : 0;
    const unsigned output_index = window_index * count + scalar_index;
    bucket_indexes[output_index] = zero_mask | window_mask | bucket_index_offset | bucket_index;
    base_indexes[output_index] = sign | scalar_index;
  }
}

__host__ hipError_t compute_bucket_indexes(const fd_q::storage *scalars, const unsigned windows_count, const unsigned window_bits, unsigned *bucket_indexes,
                                            unsigned *base_indexes, const unsigned count, hipStream_t stream) {
  const dim3 block_dim = count < MAX_THREADS ? count : MAX_THREADS;
  const dim3 grid_dim = (count - 1) / block_dim.x + 1;
  compute_bucket_indexes_kernel<<<grid_dim, block_dim, 0, stream>>>(scalars, windows_count, window_bits, bucket_indexes, base_indexes, count);
  return hipGetLastError();
}
#undef MAX_THREADS

#define MAX_THREADS 128
#ifndef __CUDACC_DEBUG__
__launch_bounds__(MAX_THREADS)
#endif
    __global__ void remove_zero_buckets_kernel(const unsigned *unique_bucket_indexes, unsigned *bucket_run_lengths, const unsigned *bucket_runs_count,
                                               const unsigned count) {
  constexpr unsigned highest_bit_mask = 0x80000000;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= count)
    return;
  const unsigned runs_count = *bucket_runs_count;
  const unsigned bucket_index = unique_bucket_indexes[gid];
  const bool is_zero = bucket_index & highest_bit_mask;
  if (gid >= runs_count || is_zero)
    bucket_run_lengths[gid] = 0;
}

__host__ hipError_t remove_zero_buckets(unsigned *unique_bucket_indexes, unsigned *bucket_run_lengths, const unsigned *bucket_runs_count, const unsigned count,
                                         hipStream_t stream) {
  const dim3 block_dim = count < MAX_THREADS ? count : MAX_THREADS;
  const dim3 grid_dim = (count - 1) / block_dim.x + 1;
  remove_zero_buckets_kernel<<<grid_dim, block_dim, 0, stream>>>(unique_bucket_indexes, bucket_run_lengths, bucket_runs_count, count);
  return hipGetLastError();
}
#undef MAX_THREADS

#define MAX_THREADS 32
#define MIN_BLOCKS 16
template <bool IS_FIRST>
#ifndef __CUDACC_DEBUG__
__launch_bounds__(MAX_THREADS, MIN_BLOCKS)
#endif
    __global__ void aggregate_buckets_kernel(const unsigned *__restrict__ base_indexes, const unsigned *__restrict__ bucket_run_offsets,
                                             const unsigned *__restrict__ bucket_run_lengths, const unsigned *__restrict__ bucket_indexes,
                                             const point_affine *__restrict__ bases, point_xyzz *__restrict__ buckets, const unsigned count) {
  constexpr unsigned negative_sign = 0x80000000;
  const field f = field();
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= count)
    return;
  const unsigned length = bucket_run_lengths[gid];
  if (length == 0)
    return;
  const unsigned base_indexes_offset = bucket_run_offsets[gid];
  const unsigned *indexes = base_indexes + base_indexes_offset;
  const unsigned bucket_index = bucket_indexes[gid] >> 1;
  point_xyzz bucket;
  if (IS_FIRST) {
    unsigned base_index = *indexes++;
    unsigned sign = base_index & negative_sign;
    base_index &= ~negative_sign;
    auto base = memory::load<point_affine, memory::ld_modifier::g>(bases + base_index);
    if (sign)
      base = point_affine::neg(base, f);
    bucket = point_affine::to_xyzz(base, f);
  } else {
    bucket = memory::load<point_xyzz, memory::ld_modifier::cs>(buckets + bucket_index);
  }
#pragma unroll 1
  for (unsigned i = IS_FIRST ? 1 : 0; i < length; i++) {
    unsigned base_index = *indexes++;
    unsigned sign = base_index & negative_sign;
    base_index &= ~negative_sign;
    auto base = memory::load<point_affine, memory::ld_modifier::g>(bases + base_index);
    if (sign)
      base = point_affine::neg(base, f);
    bucket = curve::add(bucket, base, f);
  }
  memory::store<point_xyzz, memory::st_modifier::cs>(buckets + bucket_index, bucket);
}

__host__ hipError_t aggregate_buckets(const bool is_first, const unsigned *base_indexes, const unsigned *bucket_run_offsets,
                                       const unsigned *bucket_run_lengths, const unsigned *bucket_indexes, const point_affine *bases, point_xyzz *buckets,
                                       const unsigned count, hipStream_t stream) {
  const dim3 block_dim = count < MAX_THREADS ? count : MAX_THREADS;
  const dim3 grid_dim = (count - 1) / block_dim.x + 1;
  auto kernel = is_first ? aggregate_buckets_kernel<true> : aggregate_buckets_kernel<false>;
  kernel<<<grid_dim, block_dim, 0, stream>>>(base_indexes, bucket_run_offsets, bucket_run_lengths, bucket_indexes, bases, buckets, count);
  return hipGetLastError();
}
#undef MAX_THREADS
#undef MIN_BLOCKS

#define MAX_THREADS 32
#ifndef __CUDACC_DEBUG__
__launch_bounds__(MAX_THREADS)
#endif
    __global__ void extract_top_buckets_kernel(point_xyzz *buckets, point_xyzz *top_buckets, const unsigned bits_count, const unsigned windows_count) {
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= windows_count)
    return;
  const unsigned bucket_index = gid << bits_count;
  top_buckets[gid] = buckets[bucket_index];
  buckets[bucket_index] = point_xyzz::point_at_infinity(fd_p());
}

__host__ hipError_t extract_top_buckets(point_xyzz *buckets, point_xyzz *top_buckets, const unsigned bits_count, const unsigned windows_count,
                                         hipStream_t stream) {
  const dim3 block_dim = windows_count < MAX_THREADS ? windows_count : MAX_THREADS;
  const dim3 grid_dim = (windows_count - 1) / block_dim.x + 1;
  extract_top_buckets_kernel<<<grid_dim, block_dim, 0, stream>>>(buckets, top_buckets, bits_count, windows_count);
  return hipGetLastError();
}
#undef MAX_THREADS

#define MAX_THREADS 32
#define MIN_BLOCKS 16
#ifndef __CUDACC_DEBUG__
__launch_bounds__(MAX_THREADS, MIN_BLOCKS)
#endif
    __global__ void split_windows_kernel(const unsigned source_window_bits_count, const unsigned source_windows_count,
                                         const point_xyzz *__restrict__ source_buckets, point_xyzz *__restrict__ target_buckets, const unsigned count) {
  const field f = field();
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= count)
    return;
  const unsigned target_window_bits_count = (source_window_bits_count + 1) >> 1;
  const unsigned target_windows_count = source_windows_count << 1;
  const unsigned target_partition_buckets_count = target_windows_count << target_window_bits_count;
  const unsigned target_partitions_count = count / target_partition_buckets_count;
  const unsigned target_partition_index = gid / target_partition_buckets_count;
  const unsigned target_partition_tid = gid % target_partition_buckets_count;
  const unsigned target_window_buckets_count = 1 << target_window_bits_count;
  const unsigned target_window_index = target_partition_tid / target_window_buckets_count;
  const unsigned target_window_tid = target_partition_tid % target_window_buckets_count;
  const unsigned split_index = target_window_index & 1;
  const unsigned source_window_buckets_per_target = source_window_bits_count & 1
                                                        ? split_index ? (target_window_tid >> (target_window_bits_count - 1) ? 0 : target_window_buckets_count)
                                                                      : 1 << (source_window_bits_count - target_window_bits_count)
                                                        : target_window_buckets_count;
  const unsigned source_window_index = target_window_index >> 1;
  const unsigned source_offset = source_window_index << source_window_bits_count;
  const unsigned target_shift = target_window_bits_count * split_index;
  const unsigned target_offset = target_window_tid << target_shift;
  const unsigned global_offset = source_offset + target_offset;
  const unsigned index_mask = (1 << target_shift) - 1;
  point_xyzz target_bucket = point_xyzz::point_at_infinity(f);
#pragma unroll 1
  for (unsigned i = target_partition_index; i < source_window_buckets_per_target; i += target_partitions_count) {
    const unsigned index_offset = i & index_mask | (i & ~index_mask) << target_window_bits_count;
    const unsigned load_offset = global_offset + index_offset;
    const auto source_bucket = memory::load<point_xyzz, memory::ld_modifier::g>(source_buckets + load_offset);
    target_bucket = i == target_partition_index ? source_bucket : curve::add(target_bucket, source_bucket, f);
  }
  memory::store<point_xyzz, memory::st_modifier::cs>(target_buckets + gid, target_bucket);
}

__host__ hipError_t split_windows(const unsigned source_window_bits_count, const unsigned source_windows_count, const point_xyzz *source_buckets,
                                   point_xyzz *target_buckets, const unsigned count, hipStream_t stream) {
  const dim3 block_dim = count < MAX_THREADS ? count : MAX_THREADS;
  const dim3 grid_dim = (count - 1) / block_dim.x + 1;
  split_windows_kernel<<<grid_dim, block_dim, 0, stream>>>(source_window_bits_count, source_windows_count, source_buckets, target_buckets, count);
  return hipGetLastError();
}
#undef MAX_THREADS
#undef MIN_BLOCKS

#define MAX_THREADS 32
#define MIN_BLOCKS 16
#ifndef __CUDACC_DEBUG__
__launch_bounds__(MAX_THREADS, MIN_BLOCKS)
#endif
    __global__ void reduce_buckets_kernel(point_xyzz *buckets, const unsigned count) {
  const field f = field();
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= count)
    return;
  buckets += gid;
  const auto a = memory::load<point_xyzz, memory::ld_modifier::g>(buckets);
  const auto b = memory::load<point_xyzz, memory::ld_modifier::g>(buckets + count);
  const point_xyzz result = curve::add(a, b, f);
  memory::store<point_xyzz, memory::st_modifier::cs>(buckets, result);
}

__host__ hipError_t reduce_buckets(point_xyzz *buckets, const unsigned count, hipStream_t stream) {
  const dim3 block_dim = count < MAX_THREADS ? count : MAX_THREADS;
  const dim3 grid_dim = (count - 1) / block_dim.x + 1;
  reduce_buckets_kernel<<<grid_dim, block_dim, 0, stream>>>(buckets, count);
  return hipGetLastError();
}
#undef MAX_THREADS
#undef MIN_BLOCKS

#define MAX_THREADS 32
#ifndef __CUDACC_DEBUG__
__launch_bounds__(MAX_THREADS)
#endif
    __global__ void last_pass_gather_kernel(const unsigned bits_count_pass_one, const point_xyzz *__restrict__ source, const point_xyzz *top_buckets,
                                            point_jacobian *__restrict__ target, const unsigned count) {
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= count)
    return;
  const field f = field();
  const unsigned signed_bits_count_pass_one = bits_count_pass_one - 1;
  unsigned window_index = gid / bits_count_pass_one;
  unsigned window_tid = gid % bits_count_pass_one;
  point_xyzz pz;
  if (window_tid == signed_bits_count_pass_one || gid == count - 1) {
    pz = memory::load<point_xyzz, memory::ld_modifier::g>(top_buckets + window_index);
  } else {
    for (unsigned bits_count = signed_bits_count_pass_one; bits_count > 1;) {
      bits_count = (bits_count + 1) >> 1;
      window_index <<= 1;
      if (window_tid >= bits_count) {
        window_index++;
        window_tid -= bits_count;
      }
    }
    const unsigned sid = (window_index << 1) + 1;
    pz = memory::load<point_xyzz, memory::ld_modifier::g>(source + sid);
  }
  const point_jacobian pj = point_xyzz::to_jacobian(pz, f);
  memory::store<point_jacobian, memory::st_modifier::cs>(target + gid, pj);
}

__host__ hipError_t last_pass_gather(const unsigned bits_count_pass_one, const point_xyzz *source, const point_xyzz *top_buckets, point_jacobian *target,
                                      const unsigned count, hipStream_t stream) {
  const dim3 block_dim = count < MAX_THREADS ? count : MAX_THREADS;
  const dim3 grid_dim = (count - 1) / block_dim.x + 1;
  last_pass_gather_kernel<<<grid_dim, block_dim, 0, stream>>>(bits_count_pass_one, source, top_buckets, target, count);
  return hipGetLastError();
}
#undef MAX_THREADS

template <class T> __inline__ __host__ hipError_t set_kernel_attributes(T *func) {
  HANDLE_CUDA_ERROR(hipFuncSetCacheConfig(reinterpret_cast<const void*>(func), hipFuncCachePreferL1));
  HANDLE_CUDA_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(func), hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxL1));
  return hipSuccess;
}

__host__ hipError_t set_kernel_attributes() {
  HANDLE_CUDA_ERROR(set_kernel_attributes(initialize_buckets_kernel));
  HANDLE_CUDA_ERROR(set_kernel_attributes(compute_bucket_indexes_kernel));
  HANDLE_CUDA_ERROR(set_kernel_attributes(remove_zero_buckets_kernel));
  HANDLE_CUDA_ERROR(set_kernel_attributes(aggregate_buckets_kernel<false>));
  HANDLE_CUDA_ERROR(set_kernel_attributes(aggregate_buckets_kernel<true>));
  HANDLE_CUDA_ERROR(set_kernel_attributes(extract_top_buckets_kernel));
  HANDLE_CUDA_ERROR(set_kernel_attributes(split_windows_kernel));
  HANDLE_CUDA_ERROR(set_kernel_attributes(reduce_buckets_kernel));
  HANDLE_CUDA_ERROR(set_kernel_attributes(last_pass_gather_kernel));
  return hipSuccess;
}

} // namespace msm