#include "hip/hip_runtime.h"
#include "bc.cuh"
#include "common.cuh"
#include "msm.cuh"
#include "msm_bases.cuh"
#include <algorithm>
#include <hiprand.h>
#include <numeric>
#include <random>

using namespace msm;
using namespace std;
typedef fd_p fp;
typedef fd_q fq;
typedef curve::point_affine pa;
typedef curve::point_jacobian point;

__global__ void set_bases(pa *bases, unsigned *base_multipliers, const unsigned count) {
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= count)
    return;
  const unsigned base_index = base_multipliers[gid] & 0xff;
  const pa base = pa::to_montgomery(g_bases[base_index], fp());
  bases[gid] = base;
  base_multipliers[gid] = base_index + 1;
}

__global__ void multiply_scalars_kernel(fq::storage *scalars, const unsigned *base_multipliers, const unsigned count) {
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= count)
    return;
  scalars[gid] = fq::from_montgomery(fq::mul(base_multipliers[gid], scalars[gid]));
}

class msm_test : public ::testing::Test {
protected:
  void SetUp() override { ASSERT_CUDA_SUCCESS(hipDeviceReset()); }
  void TearDown() override { hipDeviceReset(); }

  void set_up(const unsigned log_max_bases_count) {
    ASSERT_CUDA_SUCCESS(msm::set_up());
    const unsigned max_bases_count = 1 << log_max_bases_count;
    base_multipliers = new unsigned[max_bases_count];
    u_bases = new pa[max_bases_count];
    ASSERT_CUDA_SUCCESS(hipMalloc(&d_bases, sizeof(pa) * max_bases_count));
    u_scalars = new fq::storage[max_bases_count];
    ASSERT_CUDA_SUCCESS(hipMalloc(&d_scalars, sizeof(fq::storage) * max_bases_count));
    ASSERT_CUDA_SUCCESS(hipHostMalloc(&h_scalars, sizeof(fq::storage) * max_bases_count));
    u_results = new point[256];
    ASSERT_CUDA_SUCCESS(hipMalloc(&d_results, sizeof(point) * 256));
    ASSERT_CUDA_SUCCESS(hipHostMalloc(&h_results, sizeof(point) * 256));
  }

  void tear_down() {
    delete[] u_bases;
    delete[] base_multipliers;
    delete[] u_scalars;
    ASSERT_CUDA_SUCCESS(hipFree(d_bases));
    ASSERT_CUDA_SUCCESS(hipFree(d_scalars));
    ASSERT_CUDA_SUCCESS(hipHostFree(h_scalars));
    delete[] u_results;
    ASSERT_CUDA_SUCCESS(hipFree(d_results));
    ASSERT_CUDA_SUCCESS(hipHostFree(h_results));
    ASSERT_CUDA_SUCCESS(msm::tear_down());
  }

  unsigned *base_multipliers{};
  pa *u_bases{};
  pa *d_bases{};
  fq::storage *u_scalars{};
  fq::storage *d_scalars{};
  fq::storage *h_scalars{};
  point *u_results{};
  point *d_results{};
  point *h_results{};

  void generate_bases(const unsigned log_count) {
    const unsigned count = 1 << log_count;
    hiprandGenerator_t gen;
    ASSERT_CURAND_SUCCESS(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    random_device rd;
    mt19937_64 eng(rd());
    uniform_int_distribution<unsigned long long> dist;
    auto seed = dist(eng);
    ASSERT_CURAND_SUCCESS(hiprandSetPseudoRandomGeneratorSeed(gen, seed));
    unsigned *d_base_multipliers;
    ASSERT_CUDA_SUCCESS(hipMalloc(&d_base_multipliers, sizeof(unsigned) * count));
    ASSERT_CURAND_SUCCESS(hiprandGenerate(gen, d_base_multipliers, count));
    unsigned blocks_count = ((count - 1) / 32) + 1;
    set_bases<<<blocks_count, 32>>>(d_bases, d_base_multipliers, count);
    ASSERT_CUDA_SUCCESS(hipGetLastError());
    ASSERT_CUDA_SUCCESS(hipMemcpy(base_multipliers, d_base_multipliers, sizeof(unsigned) * count, hipMemcpyDeviceToHost));
    ASSERT_CUDA_SUCCESS(hipMemcpy(u_bases, d_bases, sizeof(pa) * count, hipMemcpyDeviceToHost));
    ASSERT_CUDA_SUCCESS(hipFree(d_base_multipliers));
  }

  void multiply_scalars(const unsigned count) {
    unsigned blocks_count = ((count - 1) / 32) + 1;
    unsigned *d_base_multipliers;
    ASSERT_CUDA_SUCCESS(hipMalloc(&d_base_multipliers, sizeof(unsigned) * count));
    ASSERT_CUDA_SUCCESS(hipMemcpy(d_base_multipliers, base_multipliers, sizeof(unsigned) * count, hipMemcpyHostToDevice));
    multiply_scalars_kernel<<<blocks_count, 32>>>(d_scalars, d_base_multipliers, count);
    ASSERT_CUDA_SUCCESS(hipGetLastError());
    ASSERT_CUDA_SUCCESS(hipFree(d_base_multipliers));
    ASSERT_CUDA_SUCCESS(hipMemcpy(u_scalars, d_scalars, sizeof(fq::storage) * count, hipMemcpyDeviceToHost));
  }

  point compute_checksum(const unsigned count) {
    multiply_scalars(count);
    fq::storage sum{};
    for (unsigned i = 0; i < count; i++)
      sum = fq::add(sum, u_scalars[i]);
    fp d = fp();
    return curve::mul<fq>(sum, pa::to_jacobian(pa::to_montgomery(g_bases[0], d), d), d);
  }

  point compute_result(const unsigned log_count) {
    const unsigned result_bits_count = 254;
    fp d = fp();
    point sum = point::point_at_infinity(d);
    for (unsigned i = 0; i < result_bits_count; i++) {
      unsigned index = result_bits_count - i - 1;
      point bucket = u_results[index];
      sum = i == 0 ? bucket : curve::add(curve::dbl(sum, d), bucket, d);
    }
    return sum;
  }

  static bool point_eq(const point &p1, const point &p2) { return point::eq(p1, p2, fp()); }

  void generate_scalars(const unsigned log_count) const {
    ASSERT_CUDA_SUCCESS(fields_populate_random_device<fq>(d_scalars, 1 << log_count));
    ASSERT_CUDA_SUCCESS(hipMemcpy(h_scalars, d_scalars, sizeof(fq::storage) << log_count, hipMemcpyDeviceToHost));
    ASSERT_CUDA_SUCCESS(hipMemcpy(u_scalars, h_scalars, sizeof(fq::storage) << log_count, hipMemcpyHostToHost));
  }

  void verify_result(execution_configuration cfg) {
    if (cfg.results != u_results)
      ASSERT_CUDA_SUCCESS(hipMemcpy(u_results, cfg.results, sizeof(point) * 256, hipMemcpyDefault));
    point sum = compute_result(cfg.log_scalars_count);
    point checksum = compute_checksum(1 << cfg.log_scalars_count);
    ASSERT_PRED2(point_eq, sum, checksum);
  }

  static void preallocate_pool(hipMemPool_t pool, const unsigned slack) {
    size_t mem_free = 0;
    size_t mem_total = 0;
    ASSERT_CUDA_SUCCESS(hipMemGetInfo(&mem_free, &mem_total));
    void *dummy;
    size_t dummy_size = ((mem_free >> slack) - 1) << slack;
    ASSERT_CUDA_SUCCESS(hipMallocFromPoolAsync(&dummy, dummy_size, pool, nullptr));
    ASSERT_CUDA_SUCCESS(hipFreeAsync(dummy, nullptr));
    ASSERT_CUDA_SUCCESS(hipStreamSynchronize(nullptr));
  }

  void correctness(const unsigned log_count) {
    set_up(log_count);
    generate_bases(log_count);
    generate_scalars(log_count);
    hipMemPool_t mem_pool;
    ASSERT_CUDA_SUCCESS(bc::mem_pool_create(mem_pool, 0));
    hipStream_t stream;
    ASSERT_CUDA_SUCCESS(hipStreamCreate(&stream));
    execution_configuration cfg = {mem_pool, stream, d_bases, u_scalars, u_results, log_count};
    ASSERT_CUDA_SUCCESS(execute_async(cfg));
    ASSERT_CUDA_SUCCESS(hipStreamSynchronize(stream));
    verify_result(cfg);
    ASSERT_CUDA_SUCCESS(hipStreamDestroy(stream));
    ASSERT_CUDA_SUCCESS(hipMemPoolDestroy(mem_pool));
    tear_down();
  }

  void benchmark(const vector<unsigned> &log_counts, const vector<hipMemoryType> &types) {
    const unsigned max_log_count = *max_element(log_counts.begin(), log_counts.end());
    set_up(max_log_count);
    generate_bases(max_log_count);
    generate_scalars(max_log_count);
    fq::storage *scalars;
    point *results;
    hipMemPool_t mem_pool;
    ASSERT_CUDA_SUCCESS(bc::mem_pool_create(mem_pool, 0));
    preallocate_pool(mem_pool, 25);
    hipStream_t stream;
    ASSERT_CUDA_SUCCESS(hipStreamCreate(&stream));
    ASSERT_CUDA_SUCCESS(execute_async({mem_pool, stream, d_bases, d_scalars, d_results, max_log_count}));
    ASSERT_CUDA_SUCCESS(hipDeviceSynchronize());
    printf("size");
    for (hipMemoryType type : types) {
      string type_label;
      switch (type) {
      case cudaMemoryTypeUnregistered:
        type_label = "pageable";
        break;
      case hipMemoryTypeHost:
        type_label = "pinned";
        break;
      case hipMemoryTypeDevice:
        type_label = "device";
        break;
      default:
        FAIL();
      }
      printf("\t%11s", type_label.c_str());
    }
    printf("\n");
    for (unsigned log_count : log_counts) {
      printf("2^%d", log_count);
      for (hipMemoryType type : types) {
        switch (type) {
        case cudaMemoryTypeUnregistered:
          scalars = u_scalars;
          results = u_results;
          break;
        case hipMemoryTypeHost:
          scalars = h_scalars;
          results = h_results;
          break;
        case hipMemoryTypeDevice:
          scalars = d_scalars;
          results = d_results;
          break;
        default:
          FAIL();
        }
        execution_configuration cfg = {mem_pool, stream, d_bases, scalars, results, log_count};
        hipEvent_t start;
        hipEvent_t end;
        ASSERT_CUDA_SUCCESS(hipEventCreate(&start));
        ASSERT_CUDA_SUCCESS(hipEventCreate(&end));
        ASSERT_CUDA_SUCCESS(hipEventRecord(start, stream));
        ASSERT_CUDA_SUCCESS(execute_async(cfg));
        ASSERT_CUDA_SUCCESS(hipEventRecord(end, stream));
        ASSERT_CUDA_SUCCESS(hipEventSynchronize(end));
        float elapsed;
        ASSERT_CUDA_SUCCESS(hipEventElapsedTime(&elapsed, start, end));
        printf("\t%8.3f ms", elapsed);
        ASSERT_CUDA_SUCCESS(hipEventDestroy(start));
        ASSERT_CUDA_SUCCESS(hipEventDestroy(end));
      }
      printf("\n");
    }
    ASSERT_CUDA_SUCCESS(hipStreamDestroy(stream));
    ASSERT_CUDA_SUCCESS(hipMemPoolDestroy(mem_pool));
    tear_down();
  }
};

TEST_F(msm_test, correctness_minimum_size) { correctness(0); }

TEST_F(msm_test, correctness_size_10) { correctness(10); }

TEST_F(msm_test, correctness_size_20) { correctness(20); }

TEST_F(msm_test, benchmark_range) {
  const unsigned min_log_count = 19;
  const unsigned max_log_count = 26;
  vector<unsigned> log_counts(max_log_count - min_log_count + 1);
  iota(log_counts.begin(), log_counts.end(), min_log_count);
  const vector<hipMemoryType> types = {hipMemoryTypeDevice, hipMemoryTypeHost, cudaMemoryTypeUnregistered};
  benchmark(log_counts, types);
}

TEST_F(msm_test, memory_requirements) {
  const unsigned min_log_count = 19;
  const unsigned max_log_count = 26;
  size_t zero = 0;
  ASSERT_CUDA_SUCCESS(hipMalloc(&d_bases, sizeof(pa) << max_log_count));
  ASSERT_CUDA_SUCCESS(fields_populate_random_device<fp>(reinterpret_cast<fp::storage *>(d_bases), 2 << max_log_count));
  hipMemPool_t pool;
  ASSERT_CUDA_SUCCESS(bc::mem_pool_create(pool, 0));
  hipStream_t stream;
  ASSERT_CUDA_SUCCESS(hipStreamCreate(&stream));
  printf("size\t%11s\t%11s\t%11s\t%11s\t%11s\n", "16 loops", "8 loops", "4 loops", "2 loops", "1 loop");
  for (unsigned i = min_log_count; i <= max_log_count; i++) {
    printf("2^%2d", i);
    execution_configuration cfg = {pool, stream, d_bases, reinterpret_cast<fq::storage *>(d_bases), reinterpret_cast<point_jacobian *>(d_bases), i};
    cfg.force_max_chunk_size = true;
    cfg.log_max_chunk_size = i - 4;
    if (execute_async(cfg) == hipSuccess) {
      size_t used_mem;
      ASSERT_CUDA_SUCCESS(hipMemPoolGetAttribute(pool, hipMemPoolAttrUsedMemHigh, &used_mem));
      printf("\t%11zu", used_mem);
    } else
      printf("\t%11s", "N/A");
    ASSERT_CUDA_SUCCESS(hipMemPoolSetAttribute(pool, hipMemPoolAttrUsedMemHigh, &zero));
    cfg.log_max_chunk_size = i - 3;
    if (execute_async(cfg) == hipSuccess) {
      size_t used_mem;
      ASSERT_CUDA_SUCCESS(hipMemPoolGetAttribute(pool, hipMemPoolAttrUsedMemHigh, &used_mem));
      printf("\t%11zu", used_mem);
    } else
      printf("\t%11s", "N/A");
    ASSERT_CUDA_SUCCESS(hipMemPoolSetAttribute(pool, hipMemPoolAttrUsedMemHigh, &zero));
    cfg.log_max_chunk_size = i - 2;
    if (execute_async(cfg) == hipSuccess) {
      size_t used_mem;
      ASSERT_CUDA_SUCCESS(hipMemPoolGetAttribute(pool, hipMemPoolAttrUsedMemHigh, &used_mem));
      printf("\t%11zu", used_mem);
    } else
      printf("\t%11s", "N/A");
    ASSERT_CUDA_SUCCESS(hipMemPoolSetAttribute(pool, hipMemPoolAttrUsedMemHigh, &zero));
    cfg.log_max_chunk_size = i - 1;
    if (execute_async(cfg) == hipSuccess) {
      size_t used_mem;
      ASSERT_CUDA_SUCCESS(hipMemPoolGetAttribute(pool, hipMemPoolAttrUsedMemHigh, &used_mem));
      printf("\t%11zu", used_mem);
    } else
      printf("\t%11s", "N/A");
    ASSERT_CUDA_SUCCESS(hipMemPoolSetAttribute(pool, hipMemPoolAttrUsedMemHigh, &zero));
    cfg.log_max_chunk_size = i - 0;
    if (execute_async(cfg) == hipSuccess) {
      size_t used_mem;
      ASSERT_CUDA_SUCCESS(hipMemPoolGetAttribute(pool, hipMemPoolAttrUsedMemHigh, &used_mem));
      printf("\t%11zu", used_mem);
    } else
      printf("\t%11s", "N/A");
    ASSERT_CUDA_SUCCESS(hipMemPoolSetAttribute(pool, hipMemPoolAttrUsedMemHigh, &zero));
    printf("\n");
  }
  ASSERT_CUDA_SUCCESS(hipStreamDestroy(stream));
  ASSERT_CUDA_SUCCESS(hipMemPoolDestroy(pool));
  ASSERT_CUDA_SUCCESS(hipFree(d_bases));
}

TEST_F(msm_test, benchmark_loops) {
  const unsigned min_log_count = 19;
  const unsigned max_log_count = 26;
  set_up(max_log_count);
  generate_bases(max_log_count);
  generate_scalars(max_log_count);
  hipMemPool_t pool;
  ASSERT_CUDA_SUCCESS(bc::mem_pool_create(pool, 0));
  preallocate_pool(pool, 25);
  hipStream_t stream;
  ASSERT_CUDA_SUCCESS(hipStreamCreate(&stream));
  hipEvent_t start;
  hipEvent_t end;
  ASSERT_CUDA_SUCCESS(hipEventCreate(&start));
  ASSERT_CUDA_SUCCESS(hipEventCreate(&end));
  printf("size\t%11s\t%11s\t%11s\t%11s\t%11s\n", "16 loops", "8 loops", "4 loops", "2 loops", "1 loop");
  for (unsigned i = min_log_count; i <= max_log_count; i++) {
    printf("2^%2d", i);
    execution_configuration cfg = {pool, stream, d_bases, d_scalars, d_results, i};
    cfg.force_max_chunk_size = true;
    cfg.log_max_chunk_size = i - 4;
    ASSERT_CUDA_SUCCESS(hipEventRecord(start, stream));
    if (execute_async(cfg) == hipSuccess) {
      ASSERT_CUDA_SUCCESS(hipEventRecord(end, stream));
      ASSERT_CUDA_SUCCESS(hipEventSynchronize(end));
      float elapsed;
      ASSERT_CUDA_SUCCESS(hipEventElapsedTime(&elapsed, start, end));
      printf("\t%8.3f ms", elapsed);
    } else
      printf("\t%11s", "N/A");
    cfg.log_max_chunk_size = i - 3;
    ASSERT_CUDA_SUCCESS(hipEventRecord(start, stream));
    if (execute_async(cfg) == hipSuccess) {
      ASSERT_CUDA_SUCCESS(hipEventRecord(end, stream));
      ASSERT_CUDA_SUCCESS(hipEventSynchronize(end));
      float elapsed;
      ASSERT_CUDA_SUCCESS(hipEventElapsedTime(&elapsed, start, end));
      printf("\t%8.3f ms", elapsed);
    } else
      printf("\t%11s", "N/A");
    cfg.log_max_chunk_size = i - 2;
    ASSERT_CUDA_SUCCESS(hipEventRecord(start, stream));
    if (execute_async(cfg) == hipSuccess) {
      ASSERT_CUDA_SUCCESS(hipEventRecord(end, stream));
      ASSERT_CUDA_SUCCESS(hipEventSynchronize(end));
      float elapsed;
      ASSERT_CUDA_SUCCESS(hipEventElapsedTime(&elapsed, start, end));
      printf("\t%8.3f ms", elapsed);
    } else
      printf("\t%11s", "N/A");
    cfg.log_max_chunk_size = i - 1;
    ASSERT_CUDA_SUCCESS(hipEventRecord(start, stream));
    if (execute_async(cfg) == hipSuccess) {
      ASSERT_CUDA_SUCCESS(hipEventRecord(end, stream));
      ASSERT_CUDA_SUCCESS(hipEventSynchronize(end));
      float elapsed;
      ASSERT_CUDA_SUCCESS(hipEventElapsedTime(&elapsed, start, end));
      printf("\t%8.3f ms", elapsed);
    } else
      printf("\t%11s", "N/A");
    cfg.log_max_chunk_size = i - 0;
    ASSERT_CUDA_SUCCESS(hipEventRecord(start, stream));
    if (execute_async(cfg) == hipSuccess) {
      ASSERT_CUDA_SUCCESS(hipEventRecord(end, stream));
      ASSERT_CUDA_SUCCESS(hipEventSynchronize(end));
      float elapsed;
      ASSERT_CUDA_SUCCESS(hipEventElapsedTime(&elapsed, start, end));
      printf("\t%8.3f ms", elapsed);
    } else
      printf("\t%11s", "N/A");
    printf("\n");
  }
  ASSERT_CUDA_SUCCESS(hipEventDestroy(start));
  ASSERT_CUDA_SUCCESS(hipEventDestroy(end));
  ASSERT_CUDA_SUCCESS(hipStreamDestroy(stream));
  ASSERT_CUDA_SUCCESS(hipMemPoolDestroy(pool));
  ASSERT_CUDA_SUCCESS(hipFree(d_bases));
}
